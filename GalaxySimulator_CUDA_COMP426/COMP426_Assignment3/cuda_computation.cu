#include "hip/hip_runtime.h"
#include "cuda_computation.cuh"
#include "SimulationConstants.h"
#include <stdio.h>

// Reset the QuadTree for a new set of computations
__global__ void reset_quadtree_kernel(float2* pos, float* mass, int* child, int num_particles, int num_nodes)
{
	int nodeID = threadIdx.x + blockIdx.x*blockDim.x;
	const int stride = blockDim.x*gridDim.x;

	while (nodeID < num_nodes)
	{
		// Reset all child node indices to -1 (Null)
		for (int i = 0; i < 4; ++i)
		{
			child[nodeID * 4 + i] = -1;
		}
		
		// Only reset mass and position for internal nodes
		if (nodeID >= num_particles)
		{
			pos[nodeID] = make_float2(0.0f, 0.0f);
			mass[nodeID] = 0.0f;
		}
		nodeID += stride;
	}
}

// Calculate acceleration on target_p due to force from this node's subtree on target_p
__global__ void compute_force_from_nodes_kernel(float2* pos, float2* acc, float* mass, int* child, float4* min_max_extents, int num_particles)
{
	int particleID = threadIdx.x + blockIdx.x*blockDim.x;
	const int stride = blockDim.x*gridDim.x;
	const int warp_groupID = threadIdx.x / WARP_SIZE;	// Which warp group this thread is part of in the block
	const int warpID = threadIdx.x % WARP_SIZE;			// Local ID of the thread within its warp
	const int stack_startIdx = MAX_DEPTH * warp_groupID;		// Index in the stack at which this warp's data starts

	// Create two stacks that keeps track of the size of a quadrant at each depth
	// and that tracks children to visit
	__shared__ float quadrant_size[MAX_DEPTH * BLOCKSIZE / WARP_SIZE];
	__shared__ int stack[MAX_DEPTH * BLOCKSIZE / WARP_SIZE];

	// Max radius of the top level quadrant that encompasses all the particles
	const float quadrant_radius = 0.5*(min_max_extents->x - min_max_extents->z);

	// The stack is initialized with the valid children of the root node
	// Every thread needs to be aware of how many valid children the root has
	// so that they can offset the stack's top pointer appropriately
	int stack_offset = -1;
	for (int i = 0; i < 4; ++i)
	{
		int root_nodeID = num_particles * 4;
		if (child[i + root_nodeID] != -1)
		{
			++stack_offset;
		}
	}

	// Compute acceleration for every particle assigned to this block
	while (particleID < num_particles) {

		// TODO: this should be returned from a sorted ID list
		// Ensuring that particles computed in the same warp are close to each other
		// will reduce warp divergence which results in a serial execution of the threads
		// could get close to a 32x speedup if the particles are properly sorted beforehand!
		int sortedIndex = particleID;

		float2 particle_pos = pos[sortedIndex];
		float2 particle_acc = make_float2(0.0f, 0.0f);

		// Initialize the stack using the first thread of the warp
		if (warpID == 0)
		{
			int childID = 0;
			for (int i = 0; i < 4; ++i)
			{
				// Init the stacks for the root node's children
				int root_nodeID = num_particles * 4;
				if (child[i + root_nodeID] != -1)
				{
					stack[stack_startIdx + childID] = child[i + root_nodeID];
					quadrant_size[stack_startIdx + childID] = quadrant_radius * quadrant_radius / THETA;
					++childID;
				}
			}
		}

		// Sync threads so that all threads in the block have the same stack
		__syncthreads();

		// While the stack is not empty for this warp
		int stack_top = stack_startIdx + stack_offset; // Keep track of where the stack's top pointer is for this 
		while (stack_top >= stack_startIdx)
		{
			// Get a node from the top of the stack
			int nodeID = stack[stack_top];
			// The size of a quadrant in the next depth level will be 1/4th of current one
			float next_quadrant_size = 0.25*quadrant_size[stack_top];

			// Compute acceleration from all four child nodes of the current node
			for (int i = 0; i < 4; ++i)
			{
				int childID = child[nodeID * 4 + i];

				// Make sure child is not null
				if (childID >= 0)
				{
					float2 difference_vector = pos[childID] - particle_pos;
					float squared_dist = dot(difference_vector, difference_vector) + SOFTENER; // dx*dx + dy*dy + softener

					// Compute acceleration only if the child is a particle (ie. a leaf node) or if it meets the cutoff criterion
					if (childID < num_particles || __all(next_quadrant_size <= squared_dist))
					{
						float inv_dist = rsqrtf(squared_dist); // 1/sqrt(squared_dist)

						// The particle is far enough to approximate the node as a single point
						const float g = GRAVITATIONAL_CONSTANT * mass[childID] * inv_dist * inv_dist *inv_dist;
						particle_acc += difference_vector*g;
					}
					else
					{
						// If this is the first thread of the warp, update stacks for next depth
						if (warpID == 0)
						{
							stack[stack_top] = childID;
							quadrant_size[stack_top] = next_quadrant_size;
						}
						stack_top++;
					}
				}
				else
				{
					/**
					 The article "An Efficient CUDA Implementation of the Tree - Based Barnes Hut n-Body Algorithm" (Martin Burtscher, Keshav Pingali)
					 suggests that: If the remaining nodes will also be null if this child is null then you can early-exit using:
					 stack_top = max(stack_startIdx, stack_top -1); 
					 but in this architecture we have no guarantee that if the first child is null then the second will also be null, so this isn't implemented
					*/

				}
			}
			--stack_top;
		}

		// Update the particle's acceleration
		acc[sortedIndex] = particle_acc;
		particleID += stride;

		__syncthreads();
	}
}

__global__ void integrate_kernel(float dt, float2* pos, float2* vel, const float2* acc, int num_particles)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < num_particles)
	{
		vel[i] += acc[i] * dt;
		pos[i] += vel[i] * dt;
	}
}

// Helper function to reset array values for quadtree
hipError_t reset_quadtree_with_cuda(float2* pos, float* mass, int* child)
{
	float2 *dev_pos = 0;
	float *dev_mass = 0;
	int *dev_child = 0;

	hipError_t cudaStatus;

	// Allocate GPU buffers for three vectors (three input/output)    .
	cudaStatus = hipMalloc((void**)&dev_pos, NUM_NODES * sizeof(float2));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_mass, NUM_NODES * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_child, NUM_NODES * 4 * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors and variabels from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_pos, pos, NUM_NODES * sizeof(float2), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_mass, mass, NUM_NODES * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_child, child, NUM_NODES * 4 * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	int NUM_BLOCKS = (NUM_NODES + BLOCKSIZE - 1) / BLOCKSIZE;
	reset_quadtree_kernel <<<NUM_BLOCKS, BLOCKSIZE >>>(dev_pos, dev_mass, dev_child, NUM_PARTICLES, NUM_NODES);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "reset_quadtree_kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching reset_quadtree_kernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(pos, dev_pos, NUM_NODES * sizeof(float2), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(mass, dev_mass, NUM_NODES * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(child, dev_child, NUM_NODES * 4 * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_pos);
	hipFree(dev_mass);
	hipFree(dev_child);

	return cudaStatus;
}

// Helper function for using CUDA to integrate particles in parallel
hipError_t compute_forces_and_integrate_with_cuda(float dt, float2* pos, float2* vel, const float2* acc, float* mass, int* child, float4* min_max_extents)
{
	float2 *dev_pos = 0;
	float2 *dev_vel = 0;
	float2 *dev_acc = 0;
	float *dev_mass = 0;
	int *dev_child = 0;
	float4 *dev_min_max_extents = 0;

	hipError_t cudaStatus;

	// Allocate GPU buffers for three vectors (two input/output, one pure input)    .
	cudaStatus = hipMalloc((void**)&dev_acc, NUM_PARTICLES * sizeof(float2));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_pos, NUM_NODES * sizeof(float2));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_vel, NUM_PARTICLES * sizeof(float2));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_mass, NUM_NODES * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_child, NUM_NODES * 4 * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_min_max_extents, sizeof(float4));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors and variabels from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_pos, pos, NUM_NODES * sizeof(float2), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_vel, vel, NUM_PARTICLES * sizeof(float2), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_acc, acc, NUM_PARTICLES * sizeof(float2), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_mass, mass, NUM_NODES * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_child, child, NUM_NODES * 4 * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_min_max_extents, min_max_extents, sizeof(float4), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// COMPUTE FORCES FROM EACH NODE ON PARTICLES
	// Launch a kernel on the GPU with one thread for each element.
	int NUM_BLOCKS = (NUM_PARTICLES + BLOCKSIZE - 1) / BLOCKSIZE;
	compute_force_from_nodes_kernel <<<NUM_BLOCKS, BLOCKSIZE >>>(dev_pos, dev_acc, dev_mass, dev_child, dev_min_max_extents, NUM_PARTICLES);
	
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "compute_force_from_nodes_kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching compute_force kernel!\n", cudaStatus);
		goto Error;
	}

	// INTEGRATE POSITIONS AND VELOCITIES
	// Launch a kernel on the GPU with one thread for each element.
	NUM_BLOCKS = (NUM_PARTICLES + BLOCKSIZE - 1) / BLOCKSIZE;
	integrate_kernel<<<NUM_BLOCKS, BLOCKSIZE >>>(dt, dev_pos, dev_vel, dev_acc, NUM_PARTICLES);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "integrate_kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching integrate_kernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(pos, dev_pos, NUM_NODES * sizeof(float2), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(vel, dev_vel, NUM_PARTICLES * sizeof(float2), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_acc);
	hipFree(dev_pos);
	hipFree(dev_vel);
	hipFree(dev_mass);
	hipFree(dev_child);
	hipFree(dev_min_max_extents);

	return cudaStatus;
}