#include "hip/hip_runtime.h"
#include "cuda_computation.cuh"
#include "SimulationConstants.h"
#include <stdio.h>

const int BLOCKSIZE_HOST = 64;
__device__ const int BLOCKSIZE = 64;
__device__ const int MAX_DEPTH = 64;
__device__ const int WARP_SIZE = 32;
//__device__ const float THETA = 1.0f;
//__device__ const float GRAVITATIONAL_CONSTANT = 6.67408e-11;

/**
__global__ void build_tree_kernel(float2* pos, float* mass, int* count, int* start, int* child, int* index, float2 bottom_left, float2 top_right, int num_particles, int num_nodes)
{
	int particleID = threadIdx.x + blockIdx.x*blockDim.x;
	int stride = blockDim.x*gridDim.x;
	bool particleIsInTree = false;

	// build quadtree
	float minX;
	float maxX;
	float minY;
	float maxY;
	int childPath;
	int nodeID;

	// Add every particle assigned to this block
	while ((particleID) < num_particles) {

		// If the particle isn't already in the tree, then add it
		if (!particleIsInTree) {
			particleIsInTree = true;

			minX = bottom_left.x;
			maxX = top_right.x;
			minY = bottom_left.y;
			maxY = top_right.y;

			// Check which quadrant the particle belongs to and get the appropriate path
			nodeID = 0;
			childPath = 0;
			if (pos[particleID].x < 0.5*(minX + maxX)) {
				childPath += 1;
				maxX = 0.5*(minX + maxX);
			}
			else {
				minX = 0.5*(minX + maxX);
			}
			if (pos[particleID].y < 0.5*(minY + maxY)) {
				childPath += 2;
				maxY = 0.5*(maxY + minY);
			}
			else {
				minY = 0.5*(maxY + minY);
			}
		}
		int childIndex = child[nodeID * 4 + childPath];

		// traverse tree until we hit leaf node
		while (childIndex >= num_particles) {

			// Again, at each iteration find the appropriate quadrant/childpath to traverse
			nodeID = childIndex;
			childPath = 0;
			if (pos[particleID].x < 0.5*(minX + maxX)) {
				childPath += 1;
				maxX = 0.5*(minX + maxX);
			}
			else {
				minX = 0.5*(minX + maxX);
			}
			if (pos[particleID].y < 0.5*(minY + maxY)) {
				childPath += 2;
				maxY = 0.5*(maxY + minY);
			}
			else {
				minY = 0.5*(maxY + minY);
			}

			atomicAdd(&pos[nodeID].x, mass[particleID] * pos[particleID].x);
			atomicAdd(&pos[nodeID].y, mass[particleID] * pos[particleID].y);
			atomicAdd(&mass[nodeID], mass[particleID]);
			atomicAdd(&count[nodeID], 1);
			childIndex = child[nodeID * 4 + childPath];
		}


		if (childIndex != -2) {
			int locked = nodeID * 4 + childPath;
			// TODO: atomicCAS is deprecated, replace
			if (atomicCAS(&child[locked], childIndex, -2) == childIndex) {
				if (childIndex == -1) {
					child[locked] = particleID;
				}
				else {
					int patch = 4 * num_particles; //TODO: rename
					while (childIndex >= 0 && childIndex < num_particles) {

						int node = atomicAdd(index, 1);
						patch = min(patch, node);
						if (patch != node) {
							child[4 * nodeID + childPath] = node;
						}

						// insert old particle
						childPath = 0;
						if (pos[childIndex].x < 0.5*(minX + maxX)) {
							childPath += 1;
						}
						if (pos[childIndex].y < 0.5*(minY + maxY)) {
							childPath += 2;
						}

						//if (DEBUG) {
						//
						//	if (node >= num_nodes) {
						//		printf("%s\n", "error node index is too large!!");
						//		printf("node: %d\n", node);
						//	}
						//}

						pos[node] += mass[childIndex] * pos[childIndex];
						mass[node] += mass[childIndex];
						count[node] += count[childIndex];
						child[4 * node + childPath] = childIndex;

						start[node] = -1;


						// insert new particle
						nodeID = node;
						childPath = 0;
						if (pos[particleID].x < 0.5*(minX + maxX)) {
							childPath += 1;
							maxX = 0.5*(minX + maxX);
						}
						else {
							minX = 0.5*(minX + maxX);
						}
						if (pos[particleID].y < 0.5*(minY + maxY)) {
							childPath += 2;
							maxY = 0.5*(maxY + minY);
						}
						else {
							minY = 0.5*(maxY + minY);
						}
						pos[node] += mass[particleID] * pos[particleID];
						mass[node] += mass[particleID];
						count[node] += count[particleID];
						childIndex = child[4 * nodeID + childPath];
					}

					child[4 * nodeID + childPath] = particleID;

					__threadfence();  // we have been writing to global memory arrays (child, x, y, mass) thus need to fence

					child[locked] = patch;
				}

				// __threadfence(); // we have been writing to global memory arrays (child, x, y, mass) thus need to fence

				particleID += stride;
				particleIsInTree = false;
			}

		}

		__syncthreads(); // not strictly needed 
	}
}

**/

// Reset the QuadTree for a new set of computations
__global__ void reset_quadtree_kernel(float2* pos, float* mass, int* child, int num_particles, int num_nodes)
{
	int nodeID = threadIdx.x + blockIdx.x*blockDim.x;
	const int stride = blockDim.x*gridDim.x;

	while (nodeID < num_nodes)
	{
		// Reset all child node indices to -1 (Null)
		for (int i = 0; i < 4; ++i)
		{
			child[nodeID * 4 + i] = -1;
		}
		
		// Only reset mass and position for internal nodes
		if (nodeID >= num_particles)
		{
			pos[nodeID] = make_float2(0.0f, 0.0f);
			mass[nodeID] = 0.0f;
		}
		nodeID += stride;
	}
}

// Calculate acceleration on target_p due to force from this node's subtree on target_p
__global__ void compute_force_from_nodes_kernel(float2* pos, float2* acc, float* mass, int* child, float4* min_max_extents, int num_particles)
{
	int particleID = threadIdx.x + blockIdx.x*blockDim.x;
	const int stride = blockDim.x*gridDim.x;
	const int warp_groupID = threadIdx.x / WARP_SIZE;	// Which warp group this thread is part of in the block
	const int warpID = threadIdx.x % WARP_SIZE;			// Local ID of the thread within its warp
	const int stackID = MAX_DEPTH * warp_groupID;		// Index in the stack at which this warp's data starts

	// Create two stacks that keeps track of the size of a quadrant at each depth
	// and that tracks children to visit
	__shared__ float quadrant_size[MAX_DEPTH * BLOCKSIZE / WARP_SIZE];
	__shared__ int stack[MAX_DEPTH * BLOCKSIZE / WARP_SIZE];

	// Max radius of the top level quadrant that encompasses all the particles
	const float quadrant_radius = 0.5*(min_max_extents->x - min_max_extents->z);

	int stack_offset = -1;
	for (int i = 0; i < 4; ++i)
	{
		int root_nodeID = num_particles * 4;
		if (child[i + root_nodeID] != -1)
		{
			++stack_offset;
		}
	}

	// Compute acceleration for every particle assigned to this block
	while (particleID < num_particles) {

		int sortedIndex = particleID; // TODO: this should be returned from a sorted ID list

		float2 particle_pos = pos[sortedIndex];
		float2 particle_acc = make_float2(0.0f, 0.0f);

		// Initialize the stack using the first thread of the warp
		int stack_top = stackID + stack_offset; // Keep track of where the stack's top pointer is for this warp
		if (warpID == 0)
		{
			int childID = 0;
			for (int i = 0; i < 4; ++i)
			{
				// Init the stacks for the root node's children
				int root_nodeID = num_particles * 4;
				if (child[i + root_nodeID] != -1)
				{
					stack[stackID + childID] = child[i + root_nodeID];
					quadrant_size[stackID + childID] = quadrant_radius * quadrant_radius / THETA;
					++childID;
				}
			}
		}

		// Sync threads so that all threads in the block have the same stack
		__syncthreads();

		// While the stack is not empty for this warp
		while (stack_top >= stackID)
		{
			// Get a node from the top of the stack
			int nodeID = stack[stack_top];
			// The size of a quadrant in the next depth level will be 1/4th of current one
			float next_quadrant_size = 0.25*quadrant_size[stack_top];

			// Compute acceleration from all four child nodes of the current node
			for (int i = 0; i < 4; ++i)
			{
				int childID = child[nodeID * 4 + i];

				// Make sure child is not null
				if (childID >= 0)
				{
					float2 difference_vector = pos[childID] - particle_pos;
					float squared_dist = dot(difference_vector, difference_vector) + SOFTENER; // dx*dx + dy*dy + softener

					// Compute acceleration only if the child is a particle (ie. a leaf node) or if it meets the cutoff criterion
					if (childID < num_particles || __all(next_quadrant_size <= squared_dist))
					{
						float inv_dist = rsqrtf(squared_dist); // 1/sqrt(squared_dist)

						// The particle is far enough to approximate the node as a single point
						const float g = GRAVITATIONAL_CONSTANT * mass[childID] * inv_dist * inv_dist *inv_dist;
						particle_acc += difference_vector*g;
					}
					else
					{
						// If this is the first thread of the warp, update stacks for next depth
						if (warpID == 0)
						{
							stack[stack_top] = childID;
							quadrant_size[stack_top] = next_quadrant_size;
						}
						stack_top++;
						//__syncthreads(); // TODO:  Is this necessary?
					}
				}
				else
				{
					// TODO: If the remaining nodes will also be null if this child is null then you can early-exit
					// stack_top = max(stackID, stack_top -1); 
				}
			}
			--stack_top;
		}

		// Update the particle's acceleration
		acc[sortedIndex] = particle_acc;
		particleID += stride;

		__syncthreads();
	}
}

__global__ void integrate_kernel(float dt, float2* pos, float2* vel, const float2* acc, int num_particles)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < num_particles)
	{
		vel[i] += acc[i] * dt;
		pos[i] += vel[i] * dt;
	}
}

// Helper function to reset array values for quadtree
hipError_t reset_quadtree_with_cuda(float2* pos, float* mass, int* child)
{
	float2 *dev_pos = 0;
	float *dev_mass = 0;
	int *dev_child = 0;
	//int *dev_num_particles = 0;
	//int *dev_num_nodes = 0;

	hipError_t cudaStatus;

	// Allocate GPU buffers for three vectors (three input/output)    .
	cudaStatus = hipMalloc((void**)&dev_pos, NUM_NODES * sizeof(float2));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_mass, NUM_NODES * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_child, NUM_NODES * 4 * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	//cudaStatus = hipMalloc((void**) &dev_num_particles, sizeof(int));
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMalloc failed!");
	//	goto Error;
	//}

	//cudaStatus = hipMalloc((void**) &dev_num_nodes, sizeof(int));
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMalloc failed!");
	//	goto Error;
	//}

	// Copy input vectors and variabels from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_pos, pos, NUM_NODES * sizeof(float2), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_mass, mass, NUM_NODES * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_child, child, NUM_NODES * 4 * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	//cudaStatus = hipMemset(dev_num_particles, NUM_PARTICLES, sizeof(int));
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "cudaMemSet failed!");
	//	goto Error;
	//}

	//cudaStatus = hipMemset(dev_num_nodes, NUM_NODES, sizeof(int));
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "cudaMemSet failed!");
	//	goto Error;
	//}

	// Launch a kernel on the GPU with one thread for each element.
	int NUM_BLOCKS = (NUM_NODES + BLOCKSIZE_HOST - 1) / BLOCKSIZE_HOST;
	reset_quadtree_kernel <<<NUM_BLOCKS, BLOCKSIZE_HOST >>>(dev_pos, dev_mass, dev_child, NUM_PARTICLES, NUM_NODES);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "reset_quadtree_kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching reset_quadtree_kernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(pos, dev_pos, NUM_NODES * sizeof(float2), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(mass, dev_mass, NUM_NODES * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(child, dev_child, NUM_NODES * 4 * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_pos);
	hipFree(dev_mass);
	hipFree(dev_child);
	//hipFree(dev_num_particles);
	//hipFree(dev_num_nodes);

	return cudaStatus;
}

// Helper function for using CUDA to integrate particles in parallel
hipError_t compute_forces_and_integrate_with_cuda(float dt, float2* pos, float2* vel, const float2* acc, float* mass, int* child, float4* min_max_extents)
{
	float2 *dev_pos = 0;
	float2 *dev_vel = 0;
	float2 *dev_acc = 0;
	float *dev_mass = 0;
	int *dev_child = 0;
	//int *dev_num_particles = 0;
	//int *dev_num_nodes = 0;
	float4 *dev_min_max_extents = 0;

	hipError_t cudaStatus;

	// Allocate GPU buffers for three vectors (two input/output, one pure input)    .
	cudaStatus = hipMalloc((void**)&dev_acc, NUM_PARTICLES * sizeof(float2));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_pos, NUM_NODES * sizeof(float2));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_vel, NUM_PARTICLES * sizeof(float2));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_mass, NUM_NODES * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_child, NUM_NODES * 4 * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	//cudaStatus = hipMalloc((void**)&dev_num_particles, sizeof(int));
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMalloc failed!");
	//	goto Error;
	//}

	//cudaStatus = hipMalloc((void**)&dev_num_nodes, sizeof(int));
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMalloc failed!");
	//	goto Error;
	//}

	cudaStatus = hipMalloc((void**)&dev_min_max_extents, sizeof(float4));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors and variabels from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_pos, pos, NUM_NODES * sizeof(float2), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_vel, vel, NUM_PARTICLES * sizeof(float2), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_acc, acc, NUM_PARTICLES * sizeof(float2), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_mass, mass, NUM_NODES * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_child, child, NUM_NODES * 4 * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	//cudaStatus = hipMemset(dev_num_particles, NUM_PARTICLES, sizeof(int));
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "cudaMemSet failed!");
	//	goto Error;
	//}

	//cudaStatus = hipMemset(dev_num_nodes, NUM_NODES, sizeof(int));
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "cudaMemSet failed!");
	//	goto Error;
	//}

	cudaStatus = hipMemcpy(dev_min_max_extents, min_max_extents, sizeof(float4), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// COMPUTE FORCES FROM EACH NODE ON PARTICLES
	// Launch a kernel on the GPU with one thread for each element.
	int NUM_BLOCKS = (NUM_PARTICLES + BLOCKSIZE_HOST - 1) / BLOCKSIZE_HOST;
	compute_force_from_nodes_kernel <<<NUM_BLOCKS, BLOCKSIZE_HOST >>>(dev_pos, dev_acc, dev_mass, dev_child, dev_min_max_extents, NUM_PARTICLES);
	
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "compute_force_from_nodes_kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching compute_force kernel!\n", cudaStatus);
		goto Error;
	}

	// INTEGRATE POSITIONS AND VELOCITIES
	// Launch a kernel on the GPU with one thread for each element.
	NUM_BLOCKS = (NUM_PARTICLES + BLOCKSIZE_HOST - 1) / BLOCKSIZE_HOST;
	integrate_kernel<<<NUM_BLOCKS, BLOCKSIZE_HOST >>>(dt, dev_pos, dev_vel, dev_acc, NUM_PARTICLES);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "integrate_kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching integrate_kernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(pos, dev_pos, NUM_NODES * sizeof(float2), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(vel, dev_vel, NUM_PARTICLES * sizeof(float2), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_acc);
	hipFree(dev_pos);
	hipFree(dev_vel);
	hipFree(dev_mass);
	hipFree(dev_child);
	//hipFree(dev_num_particles);
	//hipFree(dev_num_nodes);
	hipFree(dev_min_max_extents);

	return cudaStatus;
}