#include "hip/hip_runtime.h"
#include "cuda_computation.cuh"

#include <stdio.h>

__global__ void integrate_kernel(float dt, float2* pos, float2* vel, const float2* acc)
{
	int i = threadIdx.x;
	vel[i] += acc[i] * dt;
	pos[i] += vel[i] * dt;
}

// Helper function for using CUDA to integrate particles in parallel
hipError_t integrate_with_cuda(float dt, float2* pos, float2* vel, const float2* acc, unsigned int size)
{
	float2 *dev_pos = 0;
	float2 *dev_vel = 0;
	float2 *dev_acc = 0;
	hipError_t cudaStatus;

	// Allocate GPU buffers for three vectors (two input/output, one pure input)    .
	cudaStatus = hipMalloc((void**)&dev_acc, size * sizeof(float2));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_pos, size * sizeof(float2));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_vel, size * sizeof(float2));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_pos, pos, size * sizeof(float2), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_vel, vel, size * sizeof(float2), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_acc, acc, size * sizeof(float2), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	int BLOCKSIZE = 32;
	int NUM_BLOCKS = (size + BLOCKSIZE - 1) / BLOCKSIZE;
	integrate_kernel<<<NUM_BLOCKS, BLOCKSIZE >>>(dt, dev_pos, dev_vel, dev_acc);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(pos, dev_pos, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(vel, dev_vel, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_acc);
	hipFree(dev_pos);
	hipFree(dev_vel);

	return cudaStatus;
}